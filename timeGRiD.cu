#include "hip/hip_runtime.h"
/***
nvcc -std=c++11 -o timeGRiD.exe timeGRiD.cu -gencode arch=compute_86,code=sm_86 -O3 -ftz=true -prec-div=false -prec-sqrt=false
***/

#include "util/experiment_helpers.h" // include constants and other experiment consistency helpers
#include "grid.cuh"

dim3 dimms(grid::SUGGESTED_THREADS,1,1); // all loops are single loops (all mat mult flattened into column opps)
#define GRAVITY 9.81

template <typename T, int TEST_ITERS>
__host__
void test(int NUM_TIMESTEPS, hipStream_t *streams, grid::robotModel<T> *d_robotModel, grid::gridData<T> *hd_data){
   	#if TEST_FOR_EQUIVALENCE
		printf("q,qd,u\n");
		printMat<T,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
		printMat<T,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
		printMat<T,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[2*grid::NUM_JOINTS],1);

		grid::inverse_dynamics<T,false,true>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
		grid::direct_minv<T,true>(hd_data,d_robotModel,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
		grid::forward_dynamics<T>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
		grid::inverse_dynamics_gradient<T,false,true>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
		grid::forward_dynamics_gradient<T,false>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);

		printf("c\n");
		printMat<T,1,grid::NUM_JOINTS>(hd_data->h_c,1);

		printf("Minv\n");
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(hd_data->h_Minv,grid::NUM_JOINTS);

		printf("qdd\n");
		printMat<T,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);

		printf("dc_dq\n");
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(hd_data->h_dc_du,grid::NUM_JOINTS);

		printf("dc_dqd\n");
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_dc_du[grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);

		printf("df_dq\n");
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(hd_data->h_df_du,grid::NUM_JOINTS);

		printf("df_dqd\n");
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_df_du[grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);
		
   	#else
		// Setup timer
	   	struct timespec start, end;
	   	std::vector<double> times = {};

		if(NUM_TIMESTEPS == 1){
			// first one is done twice to wake up the GPU and get it up to full speed
			grid::inverse_dynamics_single_timing<T,false,true>(hd_data,d_robotModel,GRAVITY,TEST_ITERS,dim3(1,1,1),dimms,streams);
    		grid::inverse_dynamics_single_timing<T,false,true>(hd_data,d_robotModel,GRAVITY,TEST_ITERS,dim3(1,1,1),dimms,streams);

    		grid::direct_minv_single_timing<T,true>(hd_data,d_robotModel,TEST_ITERS,dim3(1,1,1),dimms,streams);

    		grid::forward_dynamics_single_timing<T>(hd_data,d_robotModel,GRAVITY,TEST_ITERS,dim3(1,1,1),dimms,streams);

    		grid::inverse_dynamics_gradient_single_timing<T,false,true>(hd_data,d_robotModel,GRAVITY,TEST_ITERS,dim3(1,1,1),dimms,streams);

    		grid::forward_dynamics_gradient_single_timing<T,false>(hd_data,d_robotModel,GRAVITY,TEST_ITERS,dim3(1,1,1),dimms,streams);
		}
		else{
			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::inverse_dynamics<T,false,true>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: ID WITH MEMORY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::inverse_dynamics_compute_only<T,false,true>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: ID COMPUTE ONLY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::direct_minv<T,true>(hd_data,d_robotModel,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: Minv WITH MEMORY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::direct_minv_compute_only<T,true>(hd_data,d_robotModel,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: Minv COMPUTE ONLY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::forward_dynamics<T>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: FD WITH MEMORY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::forward_dynamics_compute_only<T>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: FD COMPUTE ONLY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::inverse_dynamics_gradient<T,false,true>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: ID_DU WITH MEMORY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::inverse_dynamics_gradient_compute_only<T,false,true>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: ID_DU COMPUTE ONLY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::forward_dynamics_gradient<T,false>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms,streams);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: FD_DU WITH MEMORY: ",NUM_TIMESTEPS); printStats(&times); times.clear();

			for(int iter = 0; iter < TEST_ITERS; iter++){
				clock_gettime(CLOCK_MONOTONIC,&start);
				grid::forward_dynamics_gradient_compute_only<T,false>(hd_data,d_robotModel,GRAVITY,NUM_TIMESTEPS,dim3(NUM_TIMESTEPS,1,1),dimms);
				clock_gettime(CLOCK_MONOTONIC,&end);
				times.push_back(time_delta_us_timespec(start,end));
			}
			printf("[N:%d]: FD_DU COMPUTE ONLY: ",NUM_TIMESTEPS); printStats(&times); times.clear();
		}
	#endif
}

template<typename T, int TEST_ITERS>
void run_all_tests(bool floating_base){
	// allocate memory for max of what we need
	const int MAX_TIMESTEPS = 256;
	hipStream_t *streams = grid::init_grid<T>();
	grid::robotModel<T> *d_robotModel = grid::init_robotModel<T>();
	grid::gridData<T> *hd_data = grid::init_gridData<T,MAX_TIMESTEPS>();

	// load q,qd,u
	for(int k = 0; k < MAX_TIMESTEPS; k++){
		for (int ind = 0; ind < grid::NUM_JOINTS + floating_base; ind++) {
			T val = getRand<double>();
			hd_data->h_q_qd_u[k*(3*grid::NUM_JOINTS+floating_base) + ind] = val;
			hd_data->h_q_qd[k*(2*grid::NUM_JOINTS+floating_base) + ind] = val;
			hd_data->h_q[k*(grid::NUM_JOINTS+floating_base) + ind] = val;
		}
		for(int ind = 0; ind < grid::NUM_JOINTS; ind++){
			// get values
			T val2 = getRand<double>(); T val3 = getRand<double>();
			hd_data->h_q_qd_u[k*(3*grid::NUM_JOINTS+floating_base) + grid::NUM_JOINTS + ind + floating_base] = val2;
			hd_data->h_q_qd_u[k*(3*grid::NUM_JOINTS+floating_base) + 2*grid::NUM_JOINTS + ind + floating_base] = val3;
			// load into alternate memory sizes
			hd_data->h_q_qd[k*(2*grid::NUM_JOINTS+floating_base) + grid::NUM_JOINTS + ind + floating_base] = val2;
		}
	}
	// copy values onto the GPU as default values (we will do more transfers later but this ensures things are initialized)
	gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*MAX_TIMESTEPS*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(hd_data->d_q_qd,hd_data->h_q_qd,2*grid::NUM_JOINTS*MAX_TIMESTEPS*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(hd_data->d_q,hd_data->h_q,grid::NUM_JOINTS*MAX_TIMESTEPS*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	// then run the tests
	test<T,TEST_ITERS*10>(1,streams,d_robotModel,hd_data); // more iters for single test
	#if !TEST_FOR_EQUIVALENCE
		test<T,TEST_ITERS>(16,streams,d_robotModel,hd_data);
		test<T,TEST_ITERS>(32,streams,d_robotModel,hd_data);
		test<T,TEST_ITERS>(64,streams,d_robotModel,hd_data);
		test<T,TEST_ITERS>(128,streams,d_robotModel,hd_data);
		test<T,TEST_ITERS>(256,streams,d_robotModel,hd_data);
	#endif
	
	// free all memory and exit
	grid::close_grid<T>(streams,d_robotModel,hd_data);
}

int main(int argc, const char **argv){
	bool floating_base = false;
	if (argc > 1 && argv[1][0] == 'T') {floating_base = true; printf("Floating Base = True\n");}
	else {printf("Floating Base = False\n");}
	run_all_tests<float,TEST_ITERS_GLOBAL>(floating_base); return 0;
}